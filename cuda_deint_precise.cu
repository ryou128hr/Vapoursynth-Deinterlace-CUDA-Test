#include "hip/hip_runtime.h"
#include <VapourSynth4.h>
#include <VSHelper4.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdlib>
#include <cstdint>

// =======================================================
// �v���O�C���f�[�^�\��
// =======================================================
typedef struct {
    VSNode* node;
    VSVideoInfo vi;
    int tff;   // 1 = Top field first, 0 = Bottom field first
    int mode;  // 0 = double-rate (bob), 1 = single-rate
} CudaDeintData;

// =======================================================
// CUDA �J�[�l��: �}���`�^�b�v���
// =======================================================
__global__ void deintKernel(const uint8_t* src, uint8_t* dst,
    int w, int h, size_t stride,
    int useTop, int isChroma)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;

    bool missing = (useTop ? (y % 2 == 1) : (y % 2 == 0));

    if (!missing) {
        dst[y * stride + x] = src[y * stride + x];
    }
    else {
        if (!isChroma) {
            // �P�x: 4�^�b�v (�ߖT�d��)
            int sum = 0, weight = 0;
            for (int dy = -3; dy <= 3; dy += 2) {
                int yy = y + dy;
                if (yy < 0) yy = 0;
                if (yy >= h) yy = h - 1;
                int wgt = (abs(dy) == 1) ? 4 : 1;
                sum += src[yy * stride + x] * wgt;
                weight += wgt;
            }
            dst[y * stride + x] = (sum + weight / 2) / weight;
        }
        else {
            // �F����: 2�^�b�v
            int y0 = (y > 0) ? y - 1 : 0;
            int y1 = (y + 1 < h) ? y + 1 : h - 1;
            int v0 = src[y0 * stride + x];
            int v1 = src[y1 * stride + x];
            dst[y * stride + x] = (v0 + v1 + 1) / 2;
        }
    }
}

// =======================================================
// �z�X�g�����b�p�[
// =======================================================
static void runDeintKernel(const uint8_t* sp, uint8_t* dp,
    int w, int h, int stride,
    int useTop, int isChroma)
{
    size_t frame_size = (size_t)stride * h;

    uint8_t* d_src = nullptr;
    uint8_t* d_dst = nullptr;

    hipMalloc(&d_src, frame_size);
    hipMalloc(&d_dst, frame_size);
    hipMemcpy(d_src, sp, frame_size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((w + 15) / 16, (h + 15) / 16);
    deintKernel << <blocks, threads >> > (d_src, d_dst, w, h, stride, useTop, isChroma);
    hipDeviceSynchronize();

    hipMemcpy(dp, d_dst, frame_size, hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_dst);
}

// =======================================================
// �t���[������
// =======================================================
static const VSFrame* VS_CC cudaDeintGetFrame(
    int n, int activationReason, void* instanceData, void**,
    VSFrameContext* frameCtx, VSCore* core, const VSAPI* vsapi)
{
    CudaDeintData* d = (CudaDeintData*)instanceData;

    if (activationReason == arInitial) {
        int srcN = (d->mode == 0) ? n / 2 : n;
        vsapi->requestFrameFilter(srcN, d->node, frameCtx);
        return nullptr;
    }

    if (activationReason == arAllFramesReady) {
        int srcN = (d->mode == 0) ? n / 2 : n;
        const VSFrame* src = vsapi->getFrameFilter(srcN, d->node, frameCtx);

        VSFrame* dst = vsapi->newVideoFrame(&d->vi.format,
            d->vi.width, d->vi.height,
            src, core);

        bool useTop = (d->mode == 0)
            ? ((n % 2 == 0) == (d->tff == 1))
            : (d->tff == 1);

        for (int plane = 0; plane < d->vi.format.numPlanes; plane++) {
            int w = vsapi->getFrameWidth(src, plane);
            int h = vsapi->getFrameHeight(src, plane);
            int stride = vsapi->getStride(src, plane);

            const uint8_t* sp = vsapi->getReadPtr(src, plane);
            uint8_t* dp = vsapi->getWritePtr(dst, plane);

            runDeintKernel(sp, dp, w, h, stride, useTop, (plane > 0));
        }

        vsapi->freeFrame(src);
        return dst;
    }

    return nullptr;
}

// =======================================================
// Free
// =======================================================
static void VS_CC cudaDeintFree(void* instanceData, VSCore* core, const VSAPI* vsapi) {
    CudaDeintData* d = (CudaDeintData*)instanceData;
    if (d->node) vsapi->freeNode(d->node);
    free(d);
}

// =======================================================
// Create
// =======================================================
static void VS_CC cudaDeintCreate(const VSMap* in, VSMap* out, void* userData,
    VSCore* core, const VSAPI* vsapi)
{
    int err;
    CudaDeintData* d = (CudaDeintData*)malloc(sizeof(CudaDeintData));

    d->node = vsapi->mapGetNode(in, "clip", 0, &err);
    if (err) {
        vsapi->mapSetError(out, "CudaDeinterlacer: clip is required.");
        free(d);
        return;
    }

    d->vi = *vsapi->getVideoInfo(d->node);

    d->tff = (int)vsapi->mapGetInt(in, "tff", 0, &err);
    if (err) d->tff = 1;

    d->mode = (int)vsapi->mapGetInt(in, "mode", 0, &err);
    if (err) d->mode = 0;

    if (d->mode == 0) {
        d->vi.fpsNum *= 2;
        d->vi.numFrames *= 2;
    }

    VSFilterDependency deps[] = { { d->node, rpGeneral } };
    vsapi->createVideoFilter(out, "CudaDeinterlacer", &d->vi,
        cudaDeintGetFrame, cudaDeintFree,
        fmParallel, deps, 1, d, core);
}

// =======================================================
// Init
// =======================================================
VS_EXTERNAL_API(void) VapourSynthPluginInit2(VSPlugin* plugin, const VSPLUGINAPI* vspapi) {
    vspapi->configPlugin("com.example.cudadeint", "cdeint",
        "CUDA High-Quality Deinterlacer (bob/single)",
        VS_MAKE_VERSION(1, 0),
        VAPOURSYNTH_API_VERSION,
        0, plugin);

    vspapi->registerFunction("CudaDeinterlacer",
        "clip:vnode;mode:int:opt;tff:int:opt;",
        "clip:vnode;",
        cudaDeintCreate, NULL, plugin);
}
